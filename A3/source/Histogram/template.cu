#include "hip/hip_runtime.h"
#include <wb.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>

#define NUM_BINS 4096
#define BLOCK_SIZE 512 

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

__global__ void histogram(unsigned int *input, unsigned int *bins,
	unsigned int num_elements,
	unsigned int num_bins) {
	//@@ Write the kernel that computes the histogram
	//@@ Make sure to use the privitization technique
	//(hint: since NUM_BINS=4096 is larger than maximum allowed number of threads per block, 
	//be aware that threads would need to initialize more than one shared memory bin 
	//and update more than one global memory bin)

	__shared__ unsigned int s_Bins[NUM_BINS];

	int binStride = blockDim.x;
	for (int i = threadIdx.x; i < NUM_BINS; i += binStride) {
		s_Bins[i] = 0;
	}
	__syncthreads();

	int i = blockIdx.x *blockDim.x + threadIdx.x;
	for (int stride = blockDim.x * gridDim.x; i < num_elements; i += stride) {
		atomicAdd(&(s_Bins[input[i]]), 1);
	}
	__syncthreads();

	for (int j = threadIdx.x; j < NUM_BINS; j += binStride) {
		atomicAdd(&(bins[j]), s_Bins[j]);
	}
}

__global__ void saturate(unsigned int *bins, unsigned int num_bins) {
	//@@ Write the kernel that applies saturtion to counters (i.e., if the bin value is more than 127, make it equal to 127)
	int binStride = blockDim.x;
	for (int i = threadIdx.x; i < NUM_BINS; i += binStride) {
		if (bins[i] > 127) bins[i] = 127;
	}
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (unsigned int *)wbImport(wbArg_getInputFile(args, 0),
                                       &inputLength, "Integer");
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  wbLog(TRACE, "The number of bins is ", NUM_BINS);

  wbTime_start(GPU, "Allocating device memory");
  //@@ Allocate device memory here
  hipMalloc((void**)&deviceInput, inputLength * sizeof(unsigned int));
  hipMalloc((void**)&deviceBins, NUM_BINS*sizeof(unsigned int));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating device memory");

  wbTime_start(GPU, "Copying input host memory to device");
  //@@ Copy input host memory to device
  hipMemcpy(deviceInput, hostInput, inputLength * sizeof(unsigned int), hipMemcpyHostToDevice);
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input host memory to device");
	
  wbTime_start(GPU, "Clearing the bins on device");
  //@@ zero out the deviceBins using hipMemset() 
  hipMemset(deviceBins, 0, NUM_BINS*sizeof(unsigned int));
  wbTime_stop(GPU, "Clearing the bins on device");

  //@@ Initialize the grid and block dimensions here
  dim3 blockDim(BLOCK_SIZE, 1, 1);
  dim3 gridDim(((NUM_BINS - 1) / BLOCK_SIZE) + 1, 1, 1);

  wbLog(TRACE, "Launching kernel");
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Invoke kernels: first call histogram kernel and then call saturate kernel
  histogram <<<dimGrid, dimBlock>>> (deviceInput, deviceBins, inputLength, NUM_BINS);
  saturate <<<dimGrid, dimBlock>>> (deviceBins, NUM_BINS);
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output device memory to host");
  //@@ Copy output device memory to host
  hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output device memory to host");

  wbTime_start(GPU, "Freeing device memory");
  //@@ Free the device memory here
  hipFree(deviceInput);
  hipFree(deviceBins);
  wbTime_stop(GPU, "Freeing device memory");

  wbSolution(args, hostBins, NUM_BINS);

  free(hostBins);
  free(hostInput);
  return 0;
}
